#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
//#include <hip/hip_runtime.h>

#include "cublas_lay_egg.h"
#include "matrix_init_print.h"

#include <mma.h>

using namespace nvcuda;

// Tensor Core GEMM 内核
__global__ void wmma_gemm(half *a, half *b, half *c, int M, int N, int K) {

    // 声明矩阵分片
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;

    // 初始化累加器
    wmma::fill_fragment(c_frag, 0.0f);

    // 计算分片在全局内存中的位置
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // 纵向 warp i
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);            // 横向 warp j

    const int a_row = warpM * 16; //本 warp 负责的数据的 A/C 第一个行号； start_i
    const int b_col = warpN * 16; //本 warp 负责的数据的 B/C 第一个列号； start_j
    const int start_i = a_row;
    const int start_j = b_col;

    // 分块矩阵乘法
    for (int k = 0; k < K; k += 16) {
        int a_col = k; // 本轮 subA(16x16) 中，A 的第一个列号；
        int b_row = k; // 本轮 subB(16x16) 中, B 的第一个行号；

        // 检查边界
        if (a_row < M && a_col + 16 <= K) {
            wmma::load_matrix_sync(a_frag, a + a_row * K + a_col, K);// dst, 起始地址， 主维度； 自动取一个 16x16 的子矩阵。
        } else {
            // 处理边界情况
            wmma::fill_fragment(a_frag, 0.0f);
        }

        if (b_row + 16 <= K && b_col < N) {
            wmma::load_matrix_sync(b_frag, b + b_row  + b_col*K, K);
        } else {
            wmma::fill_fragment(b_frag, 0.0f);
        }

        // Tensor Core 矩阵乘加
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    // 存储结果
    if (a_row < M && b_col < N) {
        wmma::store_matrix_sync(c + start_i + start_j*M, c_frag, M, wmma::mem_col_major);
        //wmma::store_matrix_sync(c + a_row * N + b_col, c_frag, N, wmma::mem_row_major);
        //wmma::store_matrix_sync(c + a_row  + b_col*M, c_frag, M, wmma::mem_col_major);
    }
}

// 调用示例
void launch_wmma_gemm(half *A, half *B, half *C, int M, int N, int K) {
    dim3 gridDim((M + 15) / 16, (N + 15) / 16);
    dim3 blockDim(32, 4); // 128 threads per block

    wmma_gemm<<<gridDim, blockDim>>>(A, B, C, M, N, K);
}


void gemm_v02_test(int m, int n, int k,
                   half* Ah, int lda,
                   half* Bh, int ldb,
                   half* Ch, int ldc,
                   half alpha, half beta,
                   half* Dh)
{
    //1. alloc ABC_d
    half * Ad = nullptr;
    half * Bd = nullptr;
    half * Cd = nullptr;
    printf("__1000________\n");
    hipMalloc((void**)&Ad, m*lda*sizeof(half));
    hipMalloc((void**)&Bd, ldb*n*sizeof(half));
    hipMalloc((void**)&Cd, ldc*n*sizeof(half));

    //2. cpy H2D
    hipMemcpy(Ad, Ah, m*lda*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, ldb*n*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(Cd, Ch, ldc*n*sizeof(half), hipMemcpyHostToDevice);
    //3. Gemm_v01, simple cuda core gemm

printf("__01________\n");
    launch_wmma_gemm(Ad, Bd, Cd, m, n, k);
//    gemm_v01_fp16_all<<<grid_,block_>>>(m, n, k, Ad, lda, Bd, ldb, Cd, ldc, alpha, beta);
printf("##22########\n");
    //4. cpy D2H
    hipMemcpy(Dh, Cd, ldc*n*sizeof(half), hipMemcpyDeviceToHost);
    //5. free ABC_d
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}

int main()
{
#if 0
    int M = 64;
    int N = 64;
    int K = 64;
#else
    int M = 2*16;
    int N = 2*16;
    int K = 7*16;
#endif
    int lda = K;// A raw major
    int ldb = K;// B col major
    int ldc = M;// C col major

	half *A_h;
    half *B_h;
    half *C_h;
    half *D_h_tcu;
    half *D_h_cublas;


    half alpha = half(1.0);
    half beta  = half(0.0);

    A_h = (half*)malloc(M * lda * sizeof(half));
    B_h = (half*)malloc(ldb * N * sizeof(half));
    C_h = (half*)malloc(ldc * N * sizeof(half));
    D_h_tcu = (half*)malloc(ldc * N * sizeof(half));
    D_h_cublas = (half*)malloc(ldc * N * sizeof(half));

    init_matrix(A_h, lda, M, K, false);
    init_matrix(B_h, ldb, K, N, true);
    init_matrix(C_h, ldc, M, N, true);
    memcpy(D_h_tcu, C_h, M * ldc * sizeof(half));
    memcpy(D_h_cublas, C_h, M * ldc * sizeof(half));

#if 0
    printf("A_h =");
    print_matrix(A_h, lda, M, K, false);
    printf("B_h =");
    print_matrix(B_h, ldb, K, N, true);
    printf("C_h =");
    print_matrix(C_h, ldc, M, N, true);
    printf("D_h_tcu =");
    print_matrix(D_h_tcu, ldc, M, N, true);
#endif

    gemm_v02_test(M, N, K, A_h, lda, B_h, ldb, C_h, ldc, alpha, beta, D_h_tcu);
    printf("D_h_tcu = tensorCoreGemm(A, B) =\n");
    print_matrix(D_h_tcu, ldc, M, N, true);

    verify_blas(M, N, K, A_h, lda, B_h, ldb, C_h, ldc, alpha, beta, D_h_cublas);
    verify(M, N, D_h_tcu, ldc, D_h_cublas, ldc, 0.001);// relative error

    free(D_h_tcu);
    free(D_h_cublas);
    free(A_h);
    free(B_h);
    free(C_h);

    return 0;
}


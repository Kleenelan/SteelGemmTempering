#include "hip/hip_runtime.h"
#include "matrix_init_print.h"
#include "cublas_lay_egg.h"

void verify_blas(int M, int N, int K,
                 const half* Ah, int lda,
                 const half* Bh, int ldb,
                 half* Ch, int ldc,
                 const half alpha_ori,
                 const half beta_ori,
                 float* Dh)
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    half *d_A = nullptr;
    half *d_B = nullptr;
    half *d_C = nullptr;

    float* sA = (float*)malloc(lda*M * sizeof(float));
    float* sB = (float*)malloc(ldb*N * sizeof(float));
    float* sC = (float*)malloc(ldc*N * sizeof(float));

    cp_half_to_single(sA, Ah, lda*M);
    cp_half_to_single(sB, Bh, ldb*N);
    cp_half_to_single(sC, Ch, ldc*N);

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), lda*M * sizeof(float)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), ldb*N * sizeof(float)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), ldc*N * sizeof(float)));

    CUDA_CHECK(hipMemcpyAsync(d_A, sA, lda*M * sizeof(float), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, sB, ldb*N * sizeof(float), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C, sC, ldc*N * sizeof(float), hipMemcpyHostToDevice, stream));
    //CUDA_CHECK(hipStreamSynchronize(stream));//LL:: add

    float alpha = float(alpha_ori);
    float beta  = float(beta_ori);

    CUBLAS_CHECK(cublasSgemmEx(cublasH, transa, transb,
                              M, N, K,
                              &alpha,
                              d_A, HIP_R_32F, lda,
                              d_B, HIP_R_32F, ldb,
                              &beta,
                              d_C, HIP_R_32F, ldc//,
                              //HIPBLAS_COMPUTE_32F,
                            //  HIPBLAS_GEMM_DEFAULT
                            ));

    CUDA_CHECK(hipMemcpyAsync(Dh, d_C, ldc*N * sizeof(float), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

#if 0
    printf("Dh_cublasGemm =\n");
    print_matrix(Dh, ldc, M, N, true);
    printf("=====\n");
#endif

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

}

void verify(int M, int N, half* Ca, int lda, float* Cb, int ldb, double rerror)// all col
{
    long long count = 0;
    #pragma omp parallel for
    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
        {
            half a = Ca[i + j*lda];
            half b = Cb[ i + j*ldb];

            if(abs(float(a-b)/float(a)) > rerror)
            {
                printf("<%5.4f, %5.4f > ", float(a), float(b));
                count++ ;
            }
        }
    }
    printf("\n diff count = %lld\n", count);
}

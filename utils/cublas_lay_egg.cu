#include "hip/hip_runtime.h"

#include "cublas_lay_egg.h"

void verify_blas(int M, int N, int K,
                 const half* Ah, int lda,
                 const half* Bh, int ldb,
                 half* Ch, int ldc,
                 const half alpha,
                 const half beta,
                 half* Dh)
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    half *d_A = nullptr;
    half *d_B = nullptr;
    half *d_C = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), M*lda * sizeof(half)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), ldb*K * sizeof(half)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), ldc*N * sizeof(half)));

    CUDA_CHECK(hipMemcpyAsync(d_A, Ah, M*lda * sizeof(half), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, Bh, ldb*K * sizeof(half), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C, Ch, ldc*N * sizeof(half), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasGemmEx(cublasH, transa, transb,
                              M, N, K,
                              &alpha,
                              d_A, HIP_R_16F, lda,
                              d_B, HIP_R_16F, ldb,
                              &beta,
                              d_C, HIP_R_16F, ldc,
                              HIPBLAS_COMPUTE_16F,
                              HIPBLAS_GEMM_DEFAULT));

    CUDA_CHECK(hipMemcpyAsync(Dh, d_C, ldc*N * sizeof(half), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

#if 0
    printf("Dh_cublasGemm =\n");
    print_matrix(Dh, ldc, M, N, true);
    printf("=====\n");
#endif

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

}

void verify(int M, int N, half* A, int lda, half* B, int ldb, double rerror)
{
    long long count = 0;

    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
        {
            half a = A[i + j*lda];
            half b = B[ i + j*ldb];

            if(abs(float(a-b)/float(a)) > rerror)
            {
                printf("<%5.4f, %5.4f > ", float(a), float(b));
                count++ ;
            }
        }
    }
    printf("\n diff count = %lld\n", count);
}

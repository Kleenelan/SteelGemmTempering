#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
#include <omp.h>


void init_matrix(half *A, int lda, int m, int n, bool colMajor)
{
    if(colMajor)
    {
        for(int j=0; j<n; j++)
        {
            for(int i=0; i<m; i++)
            {
                half x = half(rand()*1.0f/RAND_MAX);
                A[i + j*lda] = x;
            }
        }
    }
    else
    {
        for(int i=0; i<m; i++)
        {
            for(int j=0; j<n; j++)
            {
                half x = half(rand()*1.0f/RAND_MAX);
                A[i*lda + j] = x;
            }
        }
    }
}

void print_matrix(const half *A, const int lda, const int m, const int n, bool colMajor)
{
    printf("[ ...\n");
    for(int i=0; i<m; i++)
    {

        for(int j=0; j<n; j++)
        {
            if(colMajor)
                printf(" %5.4f,", float(A[i + j*lda]));
            else
                printf(" %5.4f,", float(A[i*lda + j]));
        }
        printf(" ; ...\n");
    }
    printf("]\n");
}

void gemm_fp16_cpu(int M, int N, int K,
                   half* A, int lda,
                   half* B, int ldb,
                   half* C, int ldc,
                   half alpha, half beta)
{
    #pragma omp parallel for
    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
        {
            half sigma = half(0.0f);
            //#pragma omp parallel for
            for(int k=0; k<K; k++)
            {
                sigma += A[i*lda + k] * B[k + j*ldb];
            }
            C[i + j*ldc] = alpha*sigma + beta*C[i + j*ldc];
        }
    }
}

void verify_blas(int M, int N, int K,
                 const half* Ah, int lda,
                 const half* Bh, int ldb,
                 half* Ch, int ldc,
                 const half alpha,
                 const half beta,
                 half* Dh)
{
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    half *d_A = nullptr;
    half *d_B = nullptr;
    half *d_C = nullptr;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), M*lda * sizeof(half)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), ldb*K * sizeof(half)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), ldc*N * sizeof(half)));

    CUDA_CHECK(hipMemcpyAsync(d_A, Ah, M*lda * sizeof(half), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, Bh, ldb*K * sizeof(half), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C, Ch, ldc*N * sizeof(half), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasGemmEx(cublasH, transa, transb,
                              M, N, K,
                              &alpha,
                              d_A, HIP_R_16F, lda,
                              d_B, HIP_R_16F, ldb,
                              &beta,
                              d_C, HIP_R_16F, ldc,
                              HIPBLAS_COMPUTE_16F,
                              HIPBLAS_GEMM_DEFAULT));

    CUDA_CHECK(hipMemcpyAsync(Dh, d_C, ldc*N * sizeof(half), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

#if 0
    printf("Dh_cublasGemm =\n");
    print_matrix(Dh, ldc, M, N, true);
    printf("=====\n");
#endif

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

}


int main()
{

#if 1
    int M = 128;
    int N = 128;
    int K = 128;
#else
    int M = 7;      // \forall M > 0;
    int N = 7;      // \forall N > 0;
    int K = 4*16;   // \forall K > 0;
#endif
    int lda = K;// A raw major
    int ldb = K;// B col major
    int ldc = M;// C col major

	half *A_h = nullptr;
    half *B_h = nullptr;
    half *C_h = nullptr;
    half *D_h_cpu = nullptr;
    half *D_h_cublas = nullptr;

    half alpha = half(1.0);
    half beta  = half(0.0);

    A_h = (half*)malloc(M * lda * sizeof(half));
    B_h = (half*)malloc(ldb * N * sizeof(half));
    C_h = (half*)malloc(ldc * N * sizeof(half));
    D_h_cpu = (half*)malloc(ldc * N * sizeof(half));
    D_h_cublas = (half*)malloc(ldc * N * sizeof(half));

    init_matrix(A_h, lda, M, K, false);
    init_matrix(B_h, ldb, K, N, true);
    init_matrix(C_h, ldc, M, N, true);

    memcpy(D_h_cpu, C_h, ldc * N * sizeof(half));
    memcpy(D_h_cublas, C_h, ldc * N * sizeof(half));

#if 0
    printf("A_h =");
    print_matrix(A_h, lda, M, K, false);
    printf("B_h =");
    print_matrix(B_h, ldb, K, N, true);
    printf("C_h =");
    print_matrix(C_h, ldc, M, N, true);
#endif

    gemm_fp16_cpu(M, N, K, A_h, lda, B_h, ldb, D_h_cpu, ldc, alpha, beta);// Arow Bcol Crow Major;
    verify_blas(M, N, K, A_h, lda, B_h, ldb, C_h, ldc, alpha, beta, D_h_cublas);

#if 0
    printf("D_h_cpu = cpuGemm(A, B) =\n");
    print_matrix(D_h_cpu, ldc, M, N, true);

    printf("D_h_very=\n");
    print_matrix(D_h_cublas, ldc, M, N, true);
    printf("=====\n");
#endif

    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
        {
            half a = D_h_cpu[i + j*ldc];
            half b = D_h_cublas[ i + j*ldc];

            if(abs(float(a-b)/float(a)) > 0.01f)
            {
                printf("<%5.4f, %5.4f > ", float(a), float(b));
            }
        }
    }

    free(A_h);
    free(B_h);
    free(C_h);
    free(D_h_cpu);
    free(D_h_cublas);

    return 0;
}
